#include "hip/hip_runtime.h"
#include "utils.h"
#ifndef assert
#define assert(e)  \
	if (!(e)) { \
		printf("failed assertion `%s'\n", #e); \
		THError("aborting..."); \
	};
#endif

/*

   This file contains 2 kernels :
   - copyPixelsInSlices.
   - addPixelsInSlices.

   The primary kernel is copyPixelsInSlices : it unfolds a 3D matrix into a 2D matrix in a way that the 2D convolution (with many kernels) becomes a matrix multiplication.
   We call the resulting matrix "kernelSlices". Each row corresponds to a kW*kH*nInputPlane array.

Steps :
1) choose a pixel (pixi = blockIdx.x, pixj = blockIdx.y)
2) find which slices (coordinates (imin-imax, jmin-jmax)) will contain the pixel information
3) loop : copy the pixel information, jump to next slice (and position) by 
moving the kernelSlices pointer ptrkslices by stridej = (kH*kW - dW) * nInputPlane

detailed example : pixel (4,4), kernels of size 5*5, stride dW=1 :
- 1st slice  : top-left coordinates : (imin,jmin)  . Pixel is in coordinates (4,4, position 25) of the slice.
- 2nd slice  : top-left coordinates : (imin,jmin+1). Pixel is in coordinates (4,3, position 24) of the slice.
- 3rd slice  : top-left coordinates : (imin,jmin+2). Pixel is in coordinates (4,2, position 23) of the slice.
- 4th slice  : top-left coordinates : (imin,jmin+2). Pixel is in coordinates (4,1, position 22) of the slice.
- 5th slice  : top-left coordinates : (imin,jmin+2). Pixel is in coordinates (4,0, position 21) of the slice.
- when jmax-jmin slices have been filled, we jump to the next series of slices by 
moving ptrkslices by stridei = (((size2-jmax+jmin-1)*kH -dH)*kW  + (jmax-jmin+1)*dW)*nInputPlane
- 1st slice  : top-left coordinates : (imin+1,jmin)  . Pixel is in coordinates (3,4, position 20) of the slice.
- 2nd slice  : top-left coordinates : (imin+1,jmin+1). Pixel is in coordinates (3,3, position 19) of the slice.
- 3rd slice  : top-left coordinates : (imin+1,jmin+2). Pixel is in coordinates (3,2, position 18) of the slice.
- 4th slice  : top-left coordinates : (imin+1,jmin+2). Pixel is in coordinates (3,1, position 17) of the slice.
- 5th slice  : top-left coordinates : (imin+1,jmin+2). Pixel is in coordinates (3,0, position 16) of the slice.
- ...

In case the pixel (pixi,pixj) is in the zero-padding, we fill the slice with zeros.

addPixelsInSlices is the same, except we read the contents of the array instead of writing.

 */


__global__ void copyPixelsInSlices(float *ptrinput0, float *ptrkslices0,
		int dH, int dW, int kH, int kW, int size1, int size2, int isize1, int isize2, int nInputPlane, int padleft, int padright, int padup, int paddown, int inputstr0, int kslicesstr0, int batchsize)
{
	const int pixi=blockIdx.x;
	const int pixj=blockIdx.y;
	const int blk =blockDim.x*blockDim.y;
	const int tidx=threadIdx.x+blockDim.x*threadIdx.y;

	__shared__ int _imin, _jmin, _imax, _jmax, _stridej, _stridei, _ksliceoffset, _inputoffset;

	int imin, jmin, imax, jmax;
	int stridej, stridei, ksliceoffset, inputoffset;

	if(tidx==0)
	{
		imin=(pixi - (kH - 1) + (dH -1))/dH > 0 ? (pixi - (kH - 1) + (dH -1))/dH : 0 ;
		jmin=(pixj - (kW - 1) + (dW -1))/dW > 0 ? (pixj - (kW - 1) + (dW -1))/dW : 0 ;
		imax= pixi / dH < size1 ? pixi / dH : size1 - 1 ;
		jmax= pixj / dW < size2 ? pixj / dW : size2 - 1 ;
		stridej = (kH*kW - dW) * nInputPlane;
		stridei = (((size2-jmax+jmin-1)*kH -dH)*kW  + (jmax-jmin+1)*dW)*nInputPlane;
		ksliceoffset = ((imin * size2  + jmin) * kH * kW +  (pixi - imin * dH) * kW + (pixj - jmin*dW) ) * nInputPlane + kslicesstr0*blockIdx.z;
		inputoffset = ((pixi-padup) * isize2 + (pixj-padleft)) * nInputPlane + inputstr0*blockIdx.z;
		_imin=imin;
		_jmin=jmin;
		_imax=imax;
		_jmax=jmax;
		_stridej=stridej;
		_stridei=stridei;
		_ksliceoffset=ksliceoffset;
		_inputoffset=inputoffset;

	}

	__syncthreads();

	if(threadIdx.x==0 && threadIdx.y>0)
	{
		imin=_imin;
		jmin=_jmin;
		imax=_imax;
		jmax=_jmax;
		stridej=_stridej;
		stridei=_stridei;
		ksliceoffset=_ksliceoffset;
		inputoffset=_inputoffset;
	}

	imin=__shfl(imin, 0);
	jmin=__shfl(jmin, 0);
	imax=__shfl(imax, 0);
	jmax=__shfl(jmax, 0);
	stridej=__shfl(stridej, 0);
	stridei=__shfl(stridei, 0);
	ksliceoffset=__shfl(ksliceoffset, 0);
	inputoffset=__shfl(inputoffset, 0);

	int i;
	int j;
	int k;

	bool zeropad=pixi<padup || pixi>isize1-1+padup || pixj<padleft || pixj>isize2-1+padleft ;

	float * ptrinput    = ptrinput0 + inputoffset;
	float * ptrkslices  = ptrkslices0 + ksliceoffset;

	for(i=imin; i<imax+1; i++) {
		for(j=jmin; j<jmax+1; j++) {
			if(zeropad) 
			{
				for(k=tidx; k<nInputPlane; k+=blk) {
					ptrkslices[k]=0;
				}
			}
			else {
				for(k=tidx; k<nInputPlane; k+=blk) {
					ptrkslices[k]=ptrinput[k];
				}
			}
			ptrkslices += stridej;
		}
		ptrkslices += stridei;
	}	
}



__global__ void copyPixelsInSlicesRGB(float *ptrinput0, float *ptrkslices0,
		int dH, int dW, int kH, int kW, int size1, int size2, int isize1, int isize2, int nInputPlane, int padleft, int padright, int padup, int paddown, int inputstr0, int kslicesstr0, int batchsize)
{
	// each block does one pixel of the input image
	// each kernel slice is represented by its upper-left coordinates

	const int pixi=blockIdx.x;
	const int pixj=blockIdx.y*blockDim.y + threadIdx.y;
	const int tidx=threadIdx.x;
	const int batchindex=blockIdx.z*blockDim.z+threadIdx.z;

	int i,j;

	int imin, jmin, imax, jmax;
	int inputoffset, ksliceoffset;

	// step 1 : find which kernel slices contain the values of the pixel
	__shared__ int _imin, _jmin[32], _imax, _jmax[32], _inputoffset[32][3], _ksliceoffset[32][3]; 
	if(threadIdx.z==0)
	{
		imin=(pixi - (kH - 1) + (dH -1))/dH > 0 ? (pixi - (kH - 1) + (dH -1))/dH : 0 ;
		jmin=(pixj - (kW - 1) + (dW -1))/dW > 0 ? (pixj - (kW - 1) + (dW -1))/dW : 0 ;
		imax= pixi / dH < size1 ? pixi / dH : size1 - 1 ;
		jmax= pixj / dW < size2 ? pixj / dW : size2 - 1 ;
		if(threadIdx.x==0 && threadIdx.y==0)
		{
			_imin=imin;
			_imax=imax;
		}
		if(threadIdx.x==0)
		{
			_jmin[threadIdx.y]=jmin;
			_jmax[threadIdx.y]=jmax;
		}
		inputoffset = inputstr0*blockIdx.z*blockDim.z + ((pixi-padup) * isize2 + (pixj-padleft)) * nInputPlane ;
		ksliceoffset= kslicesstr0*blockIdx.z*blockDim.z + ((imin * size2  + jmin) * kH * kW +  (pixi - imin * dH) * kW + (pixj - jmin*dW) ) * nInputPlane;
		_inputoffset[threadIdx.y][threadIdx.x]=inputoffset;
		_ksliceoffset[threadIdx.y][threadIdx.x]=ksliceoffset;
	}

	__syncthreads();

	if(batchindex >= batchsize) return;
	if(pixj > isize2 + padleft + padright -1) return;


	if(threadIdx.z>0)
	{
		imin=_imin;
		imax=_imax;
		jmin=_jmin[threadIdx.y];
		jmax=_jmax[threadIdx.y];
		inputoffset=_inputoffset[threadIdx.y][threadIdx.x];
		ksliceoffset=_ksliceoffset[threadIdx.y][threadIdx.x];
	}

	// step 2 : move the pointers
	// this one goes to where the pixel is at
	ptrinput0   += inputoffset+inputstr0*threadIdx.z ;
	ptrkslices0 += ksliceoffset+kslicesstr0*threadIdx.z ;

	const int stridej = (kH*kW - dW) * nInputPlane;
	const int stridei = (size2*kH-dH) * kW *nInputPlane - (jmax-jmin+1) * stridej ;

	bool zeropad = pixi<padup || pixi>isize1-1+padup || pixj<padleft || pixj>isize2-1+padleft ;


	// read pixel
	// load the stuff first...
	//for (b=0; b<batchsize; b++) 
	//{
	float * ptrinput    = ptrinput0;
	float * ptrkslices  = ptrkslices0;

	float pixvalue;
	if (zeropad) 	{
		pixvalue=0;
	}
	else	{
		pixvalue=ptrinput[tidx];
	}


	//	write to memory
	for(i=imin; i<imax+1; i++) {
		for(j=jmin; j<jmax+1; j++) {
			if(zeropad) 
			{
				ptrkslices[tidx]=0;
			}
			else {
				ptrkslices[tidx]=pixvalue;
			}
			ptrkslices += stridej;
		}
		ptrkslices += stridei;
	}	
	//}
}






__global__ void addPixelsInSlices(float *ptrgradinput0, float *ptrkslices0,
		int dH, int dW, int kH, int kW, int size1, int size2, int isize1, int isize2, int nInputPlane, int padleft, int padright, int padup, int paddown, int gradinputstr0, int kslicesstr0, int batchsize)
{
	const int pixi=blockIdx.x;
	const int pixj=blockIdx.y;
	const int blk =blockDim.x*blockDim.y;
	const int tidx=threadIdx.x+blockDim.x*threadIdx.y;

	bool zeropad=pixi<padup || pixi>isize1-1+padup || pixj<padleft || pixj>isize2-1+padleft ;
	if(zeropad) return;

	__shared__ int _imin, _jmin, _imax, _jmax, _stridej, _stridei, _ksliceoffset, _gradinputoffset;
	int stridej, stridei, ksliceoffset, gradinputoffset;

	int imin;
	int jmin;
	int imax;
	int jmax;

	if(threadIdx.y==0 && threadIdx.x==0)
	{
		imin=(pixi - (kH - 1) + (dH -1))/dH > 0 ? (pixi - (kH - 1) + (dH -1))/dH : 0 ;
		jmin=(pixj - (kW - 1) + (dW -1))/dW > 0 ? (pixj - (kW - 1) + (dW -1))/dW : 0 ;
		imax= pixi / dH < size1 ? pixi / dH : size1 - 1 ;
		jmax= pixj / dW < size2 ? pixj / dW : size2 - 1 ;
		stridej = (kH*kW - dW) * nInputPlane;
		stridei = (((size2-jmax+jmin-1)*kH -dH)*kW  + (jmax-jmin+1)*dW)*nInputPlane;
		ksliceoffset = ((imin * size2  + jmin) * kH * kW +  (pixi - imin * dH) * kW + (pixj - jmin*dW) ) * nInputPlane + kslicesstr0*blockIdx.z;
		gradinputoffset = ((pixi-padup) * isize2 + (pixj-padleft)) * nInputPlane + gradinputstr0*blockIdx.z;
		_imin=imin;
		_jmin=jmin;
		_imax=imax;
		_jmax=jmax;
		_stridej=stridej;
		_stridei=stridei;
		_ksliceoffset=ksliceoffset;
		_gradinputoffset=gradinputoffset;
	}

	__syncthreads();

	if(threadIdx.x==0 && threadIdx.y>0)
	{
		imin=_imin;
		jmin=_jmin;
		imax=_imax;
		jmax=_jmax;
		stridej=_stridej;
		stridei=_stridei;
		ksliceoffset=_ksliceoffset;
		gradinputoffset=_gradinputoffset;
	}

	imin=__shfl(imin, 0);
	jmin=__shfl(jmin, 0);
	imax=__shfl(imax, 0);
	jmax=__shfl(jmax, 0);
	stridej=__shfl(stridej, 0);
	stridei=__shfl(stridei, 0);
	ksliceoffset=__shfl(ksliceoffset, 0);
	gradinputoffset=__shfl(gradinputoffset, 0);

	int i;
	int j;
	int k;

	for(k=tidx; k<nInputPlane; k+=blk) {
		float * ptrgradinput    = ptrgradinput0 + gradinputoffset;
		float * ptrkslices  		= ptrkslices0 + ksliceoffset;
		float v=0;
		for(i=imin; i<imax+1; i++) {
			for(j=jmin; j<jmax+1; j++) {
				v += ptrkslices[k];
				ptrkslices += stridej;
			}
			ptrkslices += stridei;
		}	
		ptrgradinput[k] += v;
	}
}






__global__ void copyBiasToOutputs(float *ptrbias, float *ptroutput, const int size1, const int size2, const int nOutputPlane, const int linestride, const int imstride)
{
	// each thread has a value to manage...
	//const int blk =blockDim.x;
	const int tidx=blockDim.x*blockIdx.x + threadIdx.x;
	const int tidy=blockIdx.y;
	const int tidz=blockIdx.z;	

	float val = ptrbias[tidx];
	ptroutput+= tidz*imstride + tidy*linestride;

	for(int k=0; k<size2; k++)
	{
		if(tidx<nOutputPlane) {
			ptroutput[k*nOutputPlane+tidx]=val;
		}
	}
}


void copyBiasVector(THCudaTensor* output, THCudaTensor* bias)
{
	float* ptrbias    = THCudaTensor_data(NULL, bias);
	float* ptroutput  = THCudaTensor_data(NULL, output);
	int nOutputPlane	= bias->size[0];
	int batchsize		= output->size[0];
	int size1			= output->size[1];
	int size2			= output->size[2];
	// fill output with biases
	dim3 blocksbias ((nOutputPlane+31)/32, size1, batchsize);
	dim3 threadsbias (32);
	copyBiasToOutputs<<<blocksbias, threadsbias>>>(ptrbias, ptroutput, size1, size2, nOutputPlane, output->stride[1], output->stride[0]); 

}



void sliceInput(THCudaTensor *input, THCudaTensor* kernelSlices, int kH, int kW, int dH, int dW, int padup, int paddown, int padleft, int padright)
{
	// find the size of kernelslices
	long batchsize = input->size[0];
	long isize1 = input->size[1];
	long isize2 = input->size[2];
	long nInputPlane = input->size[3];
	long size1 = (isize1 - kH + padup + paddown) / dH + 1;
	long size2 = (isize2 - kW + padleft + padright) / dW + 1;

	float* ptrkslices = THCudaTensor_data(NULL, kernelSlices);
	float* ptrinput   = THCudaTensor_data(NULL, input);

	int inputstr0=input->stride[0];
	int kslicesstr0=size1*size2*kW*kH*nInputPlane;


	//kernel unfold inputs
	if (nInputPlane ==3) 
	{
		dim3 blocksRGB (isize1 + padup + paddown, (isize2 + padleft + padright+31)/32, (batchsize+3)/4);
		dim3 threadsRGB (3,32,4);
		copyPixelsInSlicesRGB <<<blocksRGB, threadsRGB>>>(ptrinput, ptrkslices,
				dH, dW, kH, kW, size1, size2, isize1, isize2, nInputPlane, padleft, padright, padup, paddown, inputstr0, kslicesstr0, batchsize);
	}
	else 
	{
		int b_y;
		if (nInputPlane>1024) 
		{
			b_y=32;
		}
		else
		{
			b_y=(nInputPlane+31)/32;
		}
		dim3 blocks (isize1 + padup + paddown, isize2 + padleft + padright, batchsize);
		dim3 threads (32,b_y);
		copyPixelsInSlices<<<blocks, threads>>>(ptrinput, ptrkslices,
				dH, dW, kH, kW, size1, size2, isize1, isize2, nInputPlane, padleft, padright, padup, paddown, inputstr0, kslicesstr0, batchsize);
	}


}

void unsliceGradient(THCudaTensor *backwardSlices, THCudaTensor *gradInput, THCudaTensor *gradOutput, int kH, int kW, int dH, int dW, int padup, int paddown, int padleft, int padright)
{

	long batchsize = gradInput->size[0];
	long isize1 = gradInput->size[1];
	long isize2 = gradInput->size[2];
	long nInputPlane = gradInput->size[3];
	long size1 = gradOutput->size[1];
	long size2 = gradOutput->size[2];

	float* ptrbackslices = THCudaTensor_data(NULL, backwardSlices);
	float* ptrgradinput  = THCudaTensor_data(NULL, gradInput);

	int b_y;
	if (nInputPlane>1024) 
	{
		b_y=32;
	}
	else
	{
		b_y=(nInputPlane+31)/32;
	}

	dim3 blocks (isize1 + padup + paddown, isize2 + padleft + padright, batchsize);
	dim3 threads (32,b_y);

	int gradinputstr0=gradInput->stride[0];
	int kslicesstr0=size1*size2*kW*kH*nInputPlane;

	addPixelsInSlices<<<blocks, threads>>>(ptrgradinput, ptrbackslices,
			dH, dW, kH, kW, size1, size2, isize1, isize2, nInputPlane, padleft, padright, padup, paddown, gradinputstr0, kslicesstr0, batchsize);

}




static int cunxn_SpatialConvolutionUnfold_updateOutput(lua_State *L)
{
        THCState *state = getCutorchState(L);
	THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *output = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
	THCudaTensor *kernels = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
	THCudaTensor *bias = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "bias", "torch.CudaTensor");
	//  THCudaTensor *kSlices = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "kernelSlices", "torch.CudaTensor");
	long kW = luaT_getfieldcheckint(L, 1, "kW");
	long kH = luaT_getfieldcheckint(L, 1, "kH");
	long dW = luaT_getfieldcheckint(L, 1, "dW");
	long dH = luaT_getfieldcheckint(L, 1, "dH");
	long padup = luaT_getfieldcheckint(L, 1, "padtop");
	long paddown = luaT_getfieldcheckint(L, 1, "padbottom");
	long padleft = luaT_getfieldcheckint(L, 1, "padleft");
	long padright = luaT_getfieldcheckint(L, 1, "padright");
	long nOutputPlane = luaT_getfieldcheckint(L, 1, "nOutputPlane");
	long nInputPlane = luaT_getfieldcheckint(L, 1, "nInputPlane");

	// input should be contiguous already but... well.
	input = THCudaTensor_newContiguous(state, input);

	// find the size of kernelslices
	long batchsize = input->size[0];
	long isize1 = input->size[1];
	long isize2 = input->size[2];
	long size1 = (isize1 - kH + padup + paddown) / dH + 1;
	long size2 = (isize2 - kW + padleft + padright) / dW + 1;

	THCudaTensor_resize4d(state, output, batchsize, size1, size2, nOutputPlane);
	copyBiasVector(output, bias);

	// unfold conv kernels by resizing
	THCudaTensor_resize2d(state, kernels, nOutputPlane, kW*kH*nInputPlane);
	THCudaTensor_transpose(state, kernels, NULL, 0, 1);

	// in case there is not enough memory
	size_t freeMem;
	THCudaCheck(hipMemGetInfo (&freeMem, NULL));
	int nsplits=1;
	while(batchsize/nsplits*size1*size2*kW*kH*nInputPlane * 4 > freeMem) 
	{
		nsplits *= 2;
	}
	int newbatchsize=(batchsize+nsplits-1)/nsplits;
	THCudaTensor* kernelSlices = THCudaTensor_newWithSize2d(state, newbatchsize*size1*size2,kW*kH*nInputPlane);

	for(int split=0; split<nsplits; split++)
	{
		int splitsize=newbatchsize;
		if(split*newbatchsize+splitsize > batchsize)
		{
			splitsize=batchsize-split*newbatchsize;
			THCudaTensor_resize2d(state, kernelSlices, splitsize*size1*size2, kW*kH*nInputPlane);
		}
		THCudaTensor* inputSplit = THCudaTensor_newNarrow(state,input, 0, split*newbatchsize, splitsize);
		sliceInput(inputSplit, kernelSlices, kH, kW, dH, dW, padup, paddown, padleft, padright);
		THCudaTensor_free(state, inputSplit);
		THCudaTensor* outputSplit = THCudaTensor_newNarrow(state, output, 0, split*newbatchsize, splitsize);
		// put output in matrix mode
		THCudaTensor_resize2d(state, outputSplit, splitsize* size1* size2, nOutputPlane);
		//  printf("sgemm\n");
		THCudaTensor_addmm(state, outputSplit, 1, outputSplit, 1, kernelSlices, kernels);
		THCudaTensor_free(state, outputSplit);
	}




	THCudaTensor_free(state, kernelSlices); 
	THCudaTensor_transpose(state, kernels, NULL, 0, 1);
	THCudaTensor_resize4d(state, kernels, nOutputPlane, kH, kW, nInputPlane);

	// check for errors
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("error in copyPixelsInSlices: %s\n", hipGetErrorString(err));
		THError("aborting");
	}

	THCudaTensor_resize4d(state, output, batchsize, size1, size2, nOutputPlane);
	THCudaTensor_free(state, input);

	return 1;
}





static int cunxn_SpatialConvolutionUnfold_updateGradInput(lua_State *L)
{
        THCState *state = getCutorchState(L);
	THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
	long kW = luaT_getfieldcheckint(L, 1, "kW");
	long kH = luaT_getfieldcheckint(L, 1, "kH");
	long dW = luaT_getfieldcheckint(L, 1, "dW");
	long dH = luaT_getfieldcheckint(L, 1, "dH");
	long padup = luaT_getfieldcheckint(L, 1, "padtop");
	long paddown = luaT_getfieldcheckint(L, 1, "padbottom");
	long padleft = luaT_getfieldcheckint(L, 1, "padleft");
	long padright = luaT_getfieldcheckint(L, 1, "padright");
	long nOutputPlane = luaT_getfieldcheckint(L, 1, "nOutputPlane");
	long nInputPlane = luaT_getfieldcheckint(L, 1, "nInputPlane");

	THCudaTensor *kernels = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
	THCudaTensor *gradInput = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");
	THCudaTensor_resize2d(state, kernels, nOutputPlane, kW*kH*nInputPlane);

	long batchsize = input->size[0];
	long size1 = gradOutput->size[1];
	long size2 = gradOutput->size[2];

	THCudaTensor_resizeAs(state, gradInput, input);
	THCudaTensor_fill(state, gradInput, 0);

	// in case there is not enough memory
	size_t freeMem;
	THCudaCheck(hipMemGetInfo (&freeMem, NULL));
	int nsplits=1;
	while(batchsize/nsplits*size1*size2*kW*kH*nInputPlane * 4 > freeMem) 
	{
		nsplits *= 2;
	}
	int newbatchsize=(batchsize+nsplits-1)/nsplits;
	THCudaTensor* backwardSlices = THCudaTensor_newWithSize2d(state, newbatchsize*size1*size2,kW*kH*nInputPlane);


	for(int split=0; split<nsplits; split++)
	{
		int splitsize=newbatchsize;
		if(split*newbatchsize+splitsize > batchsize)
		{
			splitsize=batchsize-split*newbatchsize;
			THCudaTensor_resize2d(state, backwardSlices, splitsize*size1*size2, kW*kH*nInputPlane);
		}
		THCudaTensor* gradOutputSplit = THCudaTensor_newNarrow(state, gradOutput, 0, split*newbatchsize, splitsize);
		THCudaTensor* gradInputSplit = THCudaTensor_newNarrow(state, gradInput, 0, split*newbatchsize, splitsize);
		THCudaTensor_resize2d(state, gradOutputSplit, splitsize*size1*size2, nOutputPlane);
		// backprop gradinput into the slices
		THCudaTensor_addmm(state, backwardSlices, 0, backwardSlices, 1, gradOutputSplit, kernels);
		THCudaTensor_resize4d(state, gradOutputSplit, splitsize, size1, size2, nOutputPlane);
		unsliceGradient(backwardSlices, gradInputSplit, gradOutputSplit, kH, kW, dH, dW, padup, paddown, padleft, padright);
		THCudaTensor_free(state, gradInputSplit);
		THCudaTensor_free(state, gradOutputSplit);
	}


	// we resize gradOutput back to what it was...
	THCudaTensor_resize4d(state, gradOutput, batchsize, size1, size2, nOutputPlane);
	THCudaTensor_resize4d(state, kernels, nOutputPlane, kH, kW, nInputPlane);
	THCudaTensor_free(state, backwardSlices);

	return 1;
}



static int cunxn_SpatialConvolutionUnfold_accGradParameters(lua_State *L)
{
        THCState *state = getCutorchState(L);
	THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
	long kW = luaT_getfieldcheckint(L, 1, "kW");
	long kH = luaT_getfieldcheckint(L, 1, "kH");
	long dW = luaT_getfieldcheckint(L, 1, "dW");
	long dH = luaT_getfieldcheckint(L, 1, "dH");
	long padup = luaT_getfieldcheckint(L, 1, "padtop");
	long paddown = luaT_getfieldcheckint(L, 1, "padbottom");
	long padleft = luaT_getfieldcheckint(L, 1, "padleft");
	long padright = luaT_getfieldcheckint(L, 1, "padright");
	long nOutputPlane = luaT_getfieldcheckint(L, 1, "nOutputPlane");
	long nInputPlane = luaT_getfieldcheckint(L, 1, "nInputPlane");

	float scale = luaL_optnumber(L, 4, 1);

	// find the size of kernelslices
	long batchsize = gradOutput->size[0];

	long size1 = gradOutput->size[1];
	long size2 = gradOutput->size[2];

	THCudaTensor *gradWeight = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradWeight", "torch.CudaTensor");
	THCudaTensor *gradBias = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradBias", "torch.CudaTensor");



	float* ptrgradbias = THCudaTensor_data(state, gradBias);
	float* ptrgradoutput  = THCudaTensor_data(state, gradOutput);
	dim3 blocksgradbias (nOutputPlane+31/32);
	dim3 threadsgradbias (32,32);

	THCudaTensor_resize2d(state, gradWeight, nOutputPlane, kW*kH*nInputPlane);



	size_t freeMem;
	THCudaCheck(hipMemGetInfo (&freeMem, NULL));
	int nsplits=1;
	while(batchsize/nsplits*size1*size2*kW*kH*nInputPlane * 4 > freeMem) 
	{
		nsplits *= 2;
	}
	int newbatchsize=(batchsize+nsplits-1)/nsplits;
	THCudaTensor* kernelSlices = THCudaTensor_newWithSize2d(state, newbatchsize*size1*size2,kW*kH*nInputPlane);

	for(int split=0; split<nsplits; split++)
	{
		int splitsize=newbatchsize;
		if(split*newbatchsize+splitsize > batchsize)
		{
			splitsize=batchsize-split*newbatchsize;
			THCudaTensor_resize2d(state, kernelSlices, splitsize*size1*size2, kW*kH*nInputPlane);
		}
		THCudaTensor* gradOutputSplit = THCudaTensor_newNarrow(state, gradOutput, 0, split*newbatchsize, splitsize);
		THCudaTensor_resize2d(state, gradOutputSplit, splitsize*size1* size2, nOutputPlane);
		THCudaTensor_transpose(state, gradOutputSplit, NULL, 0, 1);
		THCudaTensor* inputSplit = THCudaTensor_newNarrow(state, input, 0, split*newbatchsize, splitsize);
		sliceInput(inputSplit, kernelSlices, kH, kW, dH, dW, padup, paddown, padleft, padright);
		THCudaTensor_addmm(state, gradWeight, 1, gradWeight, scale, gradOutputSplit, kernelSlices);
		THCudaTensor_free(state, inputSplit);
		THCudaTensor_free(state, gradOutputSplit);
	}



   THCudaTensor* gradOutTmp = THCudaTensor_newNarrow(state, gradOutput, 0, 0, batchsize);
   THCudaTensor_resize2d(state, gradOutTmp, batchsize*size1*size2, nOutputPlane);
   THCudaTensor* ones = THCudaTensor_newWithSize2d(state, 1,batchsize*size1*size2);
   THCudaTensor_fill(state, ones, 1);
   THCudaTensor_resize2d(state, gradBias, 1, nOutputPlane);
   THCudaTensor_addmm(state, gradBias, 1, gradBias, scale, ones, gradOutTmp);
   THCudaTensor_resize1d(state, gradBias, nOutputPlane);
	THCudaTensor_free(state, ones);
	THCudaTensor_free(state, gradOutTmp);

	THCudaTensor_resize4d(state, gradWeight, nOutputPlane, kH, kW, nInputPlane);

	// we resize gradOutput back to what it was...
	THCudaTensor_free(state, kernelSlices);
	
	return 1;

}

static const struct luaL_Reg cunxn_SpatialConvolutionUnfold__ [] = {
	{"SpatialConvolutionUnfold_updateOutput", cunxn_SpatialConvolutionUnfold_updateOutput},
	{"SpatialConvolutionUnfold_updateGradInput", cunxn_SpatialConvolutionUnfold_updateGradInput},
	{"SpatialConvolutionUnfold_accGradParameters", cunxn_SpatialConvolutionUnfold_accGradParameters},
	{NULL, NULL}
};

static void cunxn_SpatialConvolutionUnfold_init(lua_State *L)
{
	luaT_pushmetatable(L, "torch.CudaTensor");
	luaT_registeratname(L, cunxn_SpatialConvolutionUnfold__, "nn");
	lua_pop(L,1);
}
